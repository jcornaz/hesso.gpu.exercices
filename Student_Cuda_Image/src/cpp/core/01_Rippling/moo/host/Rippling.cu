#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Rippling.h"
#include "Device.h"

using std::cout;
using std::endl;

extern __global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

Rippling::Rippling(int w, int h, float dt) {
  assert(w == h);

  // Inputs
  this->w = w;
  this->h = h;
  this->dt = dt;

  // Tools
  this->dg = dim3(w / 32, h / 32, 1);
  this->db = dim3(32, 32, 1);
  this->t = 0;

  // Outputs
  this->title = "Rippling_Cuda";

  //print(dg, db);
  Device::assertDim(dg, db);
}

Rippling::~Rippling() {
  // rien
}

/**
 * Override
 */
void Rippling::process(uchar4* ptrDevPixels, int w, int h) {
  rippling<<<dg,db>>>(ptrDevPixels, w, h, this->t);
}


/**
 * Override
 */
void Rippling::animationStep() {
  this->t += this->dt;
}

/**
 * Override
 */
float Rippling::getAnimationPara(void) {
  return t;
}

/**
 * Override
 */
int Rippling::getW(void) {
  return w;
}

/**
 * Override
 */
int Rippling::getH(void) {
  return  h;
}

/**
 * Override
 */
string Rippling::getTitle(void) {
  return title;
}
