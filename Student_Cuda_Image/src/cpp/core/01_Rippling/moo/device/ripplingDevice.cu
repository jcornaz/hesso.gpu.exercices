#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools.h"
#include "RipplingMath.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t) {
  RipplingMath ripplingMath = RipplingMath(w, h);

  const int NB_THREADS = Indice2D::nbThread();
  const int TID = Indice2D::tid();
  const int n = w * h;
  int s = 0;
  while( s < n ) {
    int i, j;
    IndiceTools::toIJ(s, w, &i, &j);
    ripplingMath.colorIJ(&ptrDevPixels[s], i, j, t);
    s += NB_THREADS;
  }
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
