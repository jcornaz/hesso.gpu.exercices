#include "hip/hip_runtime.h"
#include <iostream>

#include "cudaTools.h"
#include "Indice1D.h"
#include "Indice2D.h"
#include "Device.h"
#include "IndiceTools.h"
#include "RipplingMath.h"

__global__ void ripplingOneToOne(uchar4* ptrDevPixels, int w, int h, float t);
__global__ void ripplingOneDimension(uchar4* ptrDevPixels, int w, int h, float t);
__global__ void ripplingTwoDimensions(uchar4* ptrDevPixels, int w, int h, float t);

__global__ void ripplingOneToOne(uchar4* ptrDevPixels, int w, int h, float t) {
  RipplingMath ripplingMath(w, h);

  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int s = j + gridDim.x * blockDim.x * (threadIdx.y + blockIdx.y * blockDim.y);

  ripplingMath.colorIJ(&ptrDevPixels[s], i, j, t);
}

__global__ void ripplingOneDimension(uchar4* ptrDevPixels, int w, int h, float t) {
  RipplingMath ripplingMath(w, h);

  const int NB_THREADS = Indice1D::nbThread();
  const int TID = Indice1D::tid();
  const int n = w * h;
  int s = TID;
  while( s < n ) {
    int i, j;
    IndiceTools::toIJ(s, w, &i, &j);
    ripplingMath.colorIJ(&ptrDevPixels[s], i, j, t);
    s += NB_THREADS;
  }
}

__global__ void ripplingTwoDimensions(uchar4* ptrDevPixels, int w, int h, float t) {
  RipplingMath ripplingMath(w, h);

  const int NB_THREADS = Indice2D::nbThread();
  const int TID = Indice2D::tid();
  const int n = w * h;
  int s = TID;
  while( s < n ) {
    int i, j;
    IndiceTools::toIJ(s, w, &i, &j);
    ripplingMath.colorIJ(&ptrDevPixels[s], i, j, t);
    s += NB_THREADS;
  }
}
