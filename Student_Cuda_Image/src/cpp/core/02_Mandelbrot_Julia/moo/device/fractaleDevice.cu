#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "Mandelbrot.h"
#include "Julia.h"
#include "DomaineMath.h"

__global__ void processMandelbrot(uchar4* ptrTabPixels, int w, int h, int n, const DomaineMath& domaineMath);
__global__ void processJulia(uchar4* ptrTabPixels, int w, int h, int n, float c1, float c2, const DomaineMath& domaineMath);

__global__ void processMandelbrot(uchar4* ptrTabPixels, int w, int h, int n, const DomaineMath& domaineMath) {
	Mandelbrot algo;
	const int WH=w*h;
  const int NB_THREADS = Indice2D::nbThread();
  const int TID = Indice2D::tid();

	int i, j;
  double x, y;

  int s = TID;
	while (s < WH) {
		IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[
  	domaineMath.toXY(i, j, &x, &y );

  	algo.colorXY(ptrTabPixels, x, y, n );

		s += NB_THREADS;
	}
}

__global__ void processJulia(uchar4* ptrTabPixels, int w, int h, int n, float c1, float c2, const DomaineMath& domaineMath) {
	Julia algo(c1, c2);
	const int WH=w*h;
  const int NB_THREADS = Indice2D::nbThread();
  const int TID = Indice2D::tid();

	int i, j;
  double x, y;

  int s = TID;
	while (s < WH) {
		IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[
  	domaineMath.toXY(i, j, &x, &y );

  	algo.colorXY(ptrTabPixels, x, y, n );

		s += NB_THREADS;
	}
}
