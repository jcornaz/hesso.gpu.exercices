#include "hip/hip_runtime.h"
#include <iostream>
#include <omp.h>

#include "FractaleMOO.h"
#include "OmpTools.h"
#include "IndiceTools.h"
#include "Device.h"

extern __global__ void processFractale(uchar4* ptrTabPixels, int w, int h, int n, const Fractale& algo, const DomaineMath& domaineMath);

FractaleMOO::FractaleMOO(int w, int h, DomaineMath* domain, Fractale* algo, int nmin, int nmax) {
	this->algo = algo;
	this->domain = domain;
  this->nmin = nmin;
  this->nmax = nmax;
  this->w = w;
  this->h = h;
  this->n = this->nmin;
	this->step = 1;

	this->dg = dim3(8, 8, 1);
	this->db = dim3(16, 16, 1);

	Device::assertDim(dg, db);
}

FractaleMOO::~FractaleMOO() {
  delete this->algo;
	delete this->domain;
}

/**
 * Override
 */
void FractaleMOO::process(uchar4* ptrDevPixels, int w, int h, const DomaineMath& domaineMath) {
	processFractale<<<dg,db>>>(ptrDevPixels, w, h, this->n, *this->algo, domaineMath );
}

DomaineMath* FractaleMOO::getDomaineMathInit() {
	return this->domain;
}

/**
 * Override
 */
void FractaleMOO::animationStep() {

	if( this->n == this->nmax ) {
		this->step = -1;
	} else if(this->n == this->nmin ) {
		this->step = 1;
	}

	this->n += this->step;
}

/**
 * Override
 */
float FractaleMOO::getAnimationPara() {
	return (float) this->n;
}

/**
 * Override
 */
int FractaleMOO::getW()	{
	return this->w;
}

/**
 * Override
 */
int FractaleMOO::getH() {
	return this->h;
}

/**
 * Override
 */
string FractaleMOO::getTitle() {
	return "Fractale_OMP";
}
