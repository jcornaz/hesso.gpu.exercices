#include "hip/hip_runtime.h"
#include "cudaType.h"
#include "Indice1D.h"
#include "IndiceTools.h"
#include "ConvolutionConstants.h"

__global__ void convertInBlackAndWhite(uchar4* ptrDevPixels, int size);
__global__ void convolution(uchar4* ptrDevPixels, uchar4* ptrDevResult, int imageWidth, int imageHeight, float* ptrDevKernel, int kernelWidth, int kernelHeight);
__global__ void computeMinMax(uchar4* ptrDevPixels, int size, int* ptrDevMin, int* ptrDevMax);
__global__ void transform(uchar4* ptrDevPixels, int size, int* ptrDevBlack, int* ptrDevWhite);

__device__ void intraThreadMinMaxReduction(int* minimumsArraySM, int* maximumsArraySM, uchar4* ptrDevPixels, int imageSize);
__device__ void intraBlockMinMaxReduction(int* minimumsArraySM, int* maximumsArraySM, int arraySize);
__device__ void interBlockMinMaxReduction(int* minimumsArraySM, int* maximumsArraySM, int* minimumResult, int* maximumResult);

__global__ void convolution(uchar4* ptrDevPixels, uchar4* ptrDevResult, int imageWidth, int imageHeight, float* ptrDevKernel, int kernelWidth, int kernelHeight) {
  const int NB_THREADS = Indice1D::nbThread();
  const int TID = Indice1D::tid();
  const int SIZE_IMAGE = imageWidth * imageHeight;
  const int SIZE_KERNEL = kernelWidth * kernelHeight;

  const int DELTA_RIGHT = kernelWidth / 2;
  const int DELTA_LEFT = kernelWidth - DELTA_RIGHT;
  const int DELTA_DOWN = kernelHeight / 2;
  const int DELTA_UP = kernelHeight - DELTA_DOWN;

  int s = TID;
  int i, j, si, sk, ik, jk;
  float sumX, sumY, sumZ;
  while (s < SIZE_IMAGE) {
    IndiceTools::toIJ(s, imageWidth, &i, &j);

    if (i - DELTA_UP >= 0 && i + DELTA_DOWN < imageHeight && j - DELTA_LEFT >= 0 && j + DELTA_RIGHT < imageWidth) {
      sumX = 0.0;
      sumY = 0.0;
      sumZ = 0.0;

      sk = 0;
      while (sk < SIZE_KERNEL) {
        IndiceTools::toIJ(sk, kernelWidth, &ik, &jk);
        si = IndiceTools::toS(imageWidth, i - DELTA_UP + ik, j - DELTA_LEFT + jk);
        sumX += ptrDevPixels[si].x * ptrDevKernel[sk];
        sumY += ptrDevPixels[si].y * ptrDevKernel[sk];
        sumZ += ptrDevPixels[si].z * ptrDevKernel[sk];
        sk++;
      }

      ptrDevResult[s].x = (int) sumX;
      ptrDevResult[s].y = (int) sumY;
      ptrDevResult[s].z = (int) sumZ;
    } else {
      ptrDevResult[s].x = 0;
      ptrDevResult[s].y = 0;
      ptrDevResult[s].z = 0;
    }

    ptrDevResult[s].w = 255;
    s += NB_THREADS;
  }
}

__global__ void convertInBlackAndWhite(uchar4* ptrDevPixels, int size) {
  const int NB_THREADS = Indice1D::nbThread();
  const int TID = Indice1D::tid();

  int s = TID;
  while (s < size) {

    char grayLevel = (ptrDevPixels[s].x + ptrDevPixels[s].y + ptrDevPixels[s].z) / 3;

    ptrDevPixels[s].x = grayLevel;
    ptrDevPixels[s].y = grayLevel;
    ptrDevPixels[s].z = grayLevel;

    s += NB_THREADS;
  }
}

__global__ void computeMinMax(uchar4* ptrDevPixels, int imageSize, int* ptrDevMin, int* ptrDevMax) {
  __shared__ int ptrDevMinimumsSM[NB_THREADS_BY_BLOCK];
  __shared__ int ptrDevMaximumsSM[NB_THREADS_BY_BLOCK];

  intraThreadMinMaxReduction(ptrDevMinimumsSM, ptrDevMaximumsSM, ptrDevPixels, imageSize);
  __syncthreads();
  intraBlockMinMaxReduction(ptrDevMinimumsSM, ptrDevMaximumsSM, NB_THREADS_BY_BLOCK);
  interBlockMinMaxReduction(ptrDevMinimumsSM, ptrDevMaximumsSM, ptrDevMin, ptrDevMax);
}

__global__ void transform(uchar4* ptrDevPixels, int size, int* ptrDevBlack, int* ptrDevWhite) {
  const int NB_THREADS = Indice1D::nbThread();
  const int TID = Indice1D::tid();

  int black = *ptrDevBlack;
  int white = *ptrDevWhite;
  int delta = abs(white - black);

  int s = TID;
  int newValue;
  while (s < size) {
    newValue = (ptrDevPixels[s].x - black) * delta + black;
    ptrDevPixels[s].x = newValue;
    ptrDevPixels[s].y = newValue;
    ptrDevPixels[s].z = newValue;
    s += NB_THREADS;
  }
}

__device__ void intraThreadMinMaxReduction(int* minimumsArraySM, int* maximumsArraySM, uchar4* ptrDevPixels, int imageSize) {
  const int NB_THREADS = Indice1D::nbThread();
  const int TID = Indice1D::tid();

  int s = TID;
  int min = 255;
  int max = 0;
  int value;
  while(s < imageSize) {
    value = ptrDevPixels[s].x;
    if (value < min) { min = value; }
    if (value > max) { max = value; }
    s += NB_THREADS;
  }

  minimumsArraySM[threadIdx.x] = min;
  maximumsArraySM[threadIdx.x] = max;
}

__device__ void intraBlockMinMaxReduction(int* minimumsArraySM, int* maximumsArraySM, int arraySize) {
  const int NB_THREADS_LOCAL = blockDim.x;
  const int TID_LOCAL = threadIdx.x;

  int n = arraySize;
  int half = arraySize / 2;
  while (half >= 1) {

    int s = TID_LOCAL;
    while (s < half) {

      if (minimumsArraySM[s + half] < minimumsArraySM[s]) {
        minimumsArraySM[s] = minimumsArraySM[s + half];
      }

      if (maximumsArraySM[s + half] > maximumsArraySM[s]) {
        maximumsArraySM[s] = maximumsArraySM[s + half];
      }

      s += NB_THREADS_LOCAL;
    }

    __syncthreads();

    n = half;
    half = n / 2;
  }
}

__device__ void interBlockMinMaxReduction(int* minimumsArraySM, int* maximumsArraySM, int* minimumResult, int* maximumResult) {
  if (threadIdx.x == 0) {
    atomicMin(minimumResult, minimumsArraySM[0]);
    atomicMax(maximumResult, maximumsArraySM[0]);
  }
}
