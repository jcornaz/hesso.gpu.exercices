#include "hip/hip_runtime.h"
#include "cudaType.h"
#include "Indice1D.h"
#include "IndiceTools.h"

#define KERNEL_WIDTH 9
#define KERNEL_SIZE 81

__constant__ float KERNEL[KERNEL_SIZE];

__global__ void convertInBlackAndWhite(uchar4* ptrDevPixels, int imageWidth, int imageHeight);
__global__ void convolution(uchar4* ptrDevPixels, uchar4* ptrDevResult, int imageWidth, int imageHeight);
__global__ void computeMinMax(uchar4* ptrDevPixels, int size, int* ptrDevMin, int* ptrDevMax);
__global__ void transform(uchar4* ptrDevPixels, int size, int* ptrDevBlack, int* ptrDevWhite);

float* getPtrDevKernel() {
  float* ptrDevKernel;
  HANDLE_ERROR(hipGetSymbolAddress((void**) &ptrDevKernel, KERNEL));
  return ptrDevKernel;
}

__global__ void convolution(uchar4* ptrDevPixels, uchar4* ptrDevResult, int imageWidth, int imageHeight) {
  const int NB_THREADS = Indice1D::nbThread();
  const int TID = Indice1D::tid();
  const int IMAGE_SIZE = imageWidth * imageHeight;
  const int HALF_KERNEL_SIZE = KERNEL_SIZE / 2;
  const int HALF_KERNEL_WIDTH = KERNEL_WIDTH / 2;

  int s = TID;
  int i, j;
  float sum;
  while (s < IMAGE_SIZE) {
    IndiceTools::toIJ(s, imageWidth, &i, &j);

    if (i - HALF_KERNEL_WIDTH >= 0 && i + HALF_KERNEL_WIDTH < imageHeight && j - HALF_KERNEL_WIDTH >= 0 && j + HALF_KERNEL_WIDTH < imageWidth) {
      sum = 0.0;

      for (int v = 1 ; v <= HALF_KERNEL_WIDTH ; v++) {
        for (int u = 1 ; u <= HALF_KERNEL_WIDTH ; u++) {
          sum += ptrDevPixels[s + v * imageWidth + u].x * KERNEL[HALF_KERNEL_SIZE + v * KERNEL_WIDTH + u];
          sum += ptrDevPixels[s - v * imageWidth + u].x * KERNEL[HALF_KERNEL_SIZE - v * KERNEL_WIDTH + u];
          sum += ptrDevPixels[s + v * imageWidth - u].x * KERNEL[HALF_KERNEL_SIZE + v * KERNEL_WIDTH - u];
          sum += ptrDevPixels[s - v * imageWidth - u].x * KERNEL[HALF_KERNEL_SIZE - v * KERNEL_WIDTH - u];
        }

        sum += ptrDevPixels[s - v * imageWidth].x * KERNEL[HALF_KERNEL_SIZE - v * KERNEL_WIDTH];
        sum += ptrDevPixels[s + v * imageWidth].x * KERNEL[HALF_KERNEL_SIZE + v * KERNEL_WIDTH];
        sum += ptrDevPixels[s + v].x * KERNEL[HALF_KERNEL_SIZE + v];
        sum += ptrDevPixels[s - v].x * KERNEL[HALF_KERNEL_SIZE - v];
      }

      sum += ptrDevPixels[s].x * KERNEL[HALF_KERNEL_SIZE];

      ptrDevResult[s].x = (int) sum;
      ptrDevResult[s].y = (int) sum;
      ptrDevResult[s].z = (int) sum;
    } else {
      ptrDevResult[s].x = 0;
      ptrDevResult[s].y = 0;
      ptrDevResult[s].z = 0;
    }

    ptrDevResult[s].w = 255;
    s += NB_THREADS;
  }
}

__global__ void convertInBlackAndWhite(uchar4* ptrDevPixels, int imageWidth, int imageHeight) {
  const int NB_THREADS = Indice1D::nbThread();
  const int TID = Indice1D::tid();
  const int N = imageWidth * imageHeight;

  int s = TID;
  while (s < N) {

    char grayLevel = (ptrDevPixels[s].x + ptrDevPixels[s].y + ptrDevPixels[s].z) / 3;

    ptrDevPixels[s].x = grayLevel;
    ptrDevPixels[s].y = grayLevel;
    ptrDevPixels[s].z = grayLevel;

    s += NB_THREADS;
  }
}

__global__ void computeMinMax(uchar4* ptrDevPixels, int size, int* ptrDevMin, int* ptrDevMax) {
  const int NB_THREADS = Indice1D::nbThread();
  const int TID = Indice1D::tid();

  // TODO
}

__global__ void transform(uchar4* ptrDevPixels, int size, int* ptrDevBlack, int* ptrDevWhite) {
  const int NB_THREADS = Indice1D::nbThread();
  const int TID = Indice1D::tid();

  // TODO
}
