#include "hip/hip_runtime.h"
#include "ConvolutionMOO.h"
#include "Device.h"
#include "OpencvTools.h"
#include "cudaType.h"

extern __global__ void convolution(uchar4* ptrDevPixels, uchar4* ptrDevResult, int imageWidth, int imageHeight, float* ptrDevKernel, int kernelWidth, int kernelHeight);
extern __global__ void transform(uchar4* ptrDevPixels, uchar4* ptrDevResult, int imageWidth, int imageHeight, int kernelWidth);
extern __global__ void convertInBlackAndWhite(uchar4* ptrDevPixels, int imageWidth, int imageHeight);

ConvolutionMOO::ConvolutionMOO(string videoPath, int kernelWidth, int kernelHeight, float* ptrKernel) {

  this->dg = dim3(64, 64, 1);
  this->db = dim3(32, 32, 1);
  Device::assertDim(dg, db);

  this->t = 0;
  this->kernelWidth = kernelWidth;
  this->kernelHeight = kernelHeight;
  this->videoCapter = new CVCaptureVideo("/media/Data/Video/autoroute.mp4");
  this->videoCapter->start();

  size_t kernelSize = sizeof(float) * kernelWidth * kernelHeight;
  HANDLE_ERROR(hipMalloc(&this->ptrDevKernel, kernelSize));
  HANDLE_ERROR(hipMalloc(&this->ptrDevImage, sizeof(uchar4) * this->videoCapter->getW() * this->videoCapter->getH()));
  HANDLE_ERROR(hipMemcpy(this->ptrDevKernel, ptrKernel, kernelSize, hipMemcpyHostToDevice));

  std::cout << this->videoCapter->getW() << ", " << this->videoCapter->getH() << std::endl;
}

ConvolutionMOO::~ConvolutionMOO() {
  this->videoCapter->stop();
  free(this->videoCapter);
  HANDLE_ERROR(hipFree(this->ptrDevKernel));
  HANDLE_ERROR(hipFree(this->ptrDevImage));
}

/**
* Call periodicly by the api
*/
void ConvolutionMOO::process(uchar4* ptrDevPixels, int w, int h) {
  Mat matRGBA(h, w, CV_8UC4);
  Mat matBGR = this->videoCapter->provideBGR();
  OpencvTools::switchRB(matRGBA, matBGR);
  uchar4* ptrImage = OpencvTools::castToUchar4(matRGBA);

  HANDLE_ERROR(hipMemcpy(this->ptrDevImage, ptrImage, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));
  
  convertInBlackAndWhite<<<dg,db>>>(this->ptrDevImage, w, h);
  convolution<<<dg,db>>>(this->ptrDevImage, ptrDevPixels, w, h, this->ptrDevKernel, this->kernelWidth, this->kernelHeight);

  HANDLE_ERROR(hipMemcpy(this->ptrDevImage, ptrDevPixels, sizeof(uchar4) * w * h, hipMemcpyDeviceToDevice));

  transform<<<dg,db>>>(this->ptrDevImage, ptrDevPixels, w, h, 3);
}

/**
* Call periodicly by the api
*/
void ConvolutionMOO::animationStep() {
  this->t++;
}

float ConvolutionMOO::getAnimationPara() {
  return this->t;
}

string ConvolutionMOO::getTitle() {
  return "Convolution";
}

int ConvolutionMOO::getW() {
  return this->videoCapter->getW();
}

int ConvolutionMOO::getH() {
  return this->videoCapter->getH();
}
