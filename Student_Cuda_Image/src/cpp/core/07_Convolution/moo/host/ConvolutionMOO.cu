#include "hip/hip_runtime.h"
#include "ConvolutionMOO.h"
#include "Device.h"
#include "OpencvTools.h"
#include "cudaType.h"

extern __global__ void convolution(uchar4* ptrDevPixels, uchar4* ptrDevResult, int imageWidth, int imageHeight, float* ptrDevKernel, int kernelWidth);
extern __global__ void transform(uchar4* ptrDevPixels, uchar4* ptrDevResult, int imageWidth, int imageHeight, int kernelWidth);
extern __global__ void convertInBlackAndWhite(uchar4* ptrDevPixels, int imageWidth, int imageHeight);

ConvolutionMOO::ConvolutionMOO(string videoPath, int kernelWidth, float* ptrKernel) {

  this->dg = dim3(1096, 1, 1);
  this->db = dim3(1024, 1, 1);
  Device::assertDim(dg, db);

  this->t = 0;
  this->kernelWidth = kernelWidth;
  this->videoCapter = new CVCaptureVideo("/media/Data/Video/autoroute.mp4");
  this->videoCapter->start();

  size_t kernelSize = sizeof(float) * kernelWidth * kernelWidth;
  HANDLE_ERROR(hipMalloc(&this->ptrDevKernel, kernelSize));
  HANDLE_ERROR(hipMalloc(&this->ptrDevImage, sizeof(uchar4) * this->videoCapter->getW() * this->videoCapter->getH()));
  HANDLE_ERROR(hipMemcpy(this->ptrDevKernel, ptrKernel, kernelSize, hipMemcpyHostToDevice));
}

ConvolutionMOO::~ConvolutionMOO() {
  this->videoCapter->stop();
  free(this->videoCapter);
  HANDLE_ERROR(hipFree(this->ptrDevKernel));
  HANDLE_ERROR(hipFree(this->ptrDevImage));
}

/**
* Call periodicly by the api
*/
void ConvolutionMOO::process(uchar4* ptrDevPixels, int w, int h) {
  Mat matRGBA(h, w, CV_8UC4);
  Mat matBGR = this->videoCapter->provideBGR();
  OpencvTools::switchRB(matRGBA, matBGR);
  uchar4* ptrImage = OpencvTools::castToUchar4(matRGBA);

  HANDLE_ERROR(hipMemcpy(this->ptrDevImage, ptrImage, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

  convertInBlackAndWhite<<<dg,db>>>(this->ptrDevImage, w, h);
  convolution<<<dg,db>>>(this->ptrDevImage, ptrDevPixels, w, h, this->ptrDevKernel, this->kernelWidth);

  HANDLE_ERROR(hipMemcpy(this->ptrDevImage, ptrDevPixels, sizeof(uchar4) * w * h, hipMemcpyDeviceToDevice));

  transform<<<dg,db>>>(this->ptrDevImage, ptrDevPixels, w, h, 3);
}

/**
* Call periodicly by the api
*/
void ConvolutionMOO::animationStep() {
  this->t++;
}

float ConvolutionMOO::getAnimationPara() {
  return this->t;
}

string ConvolutionMOO::getTitle() {
  return "Convolution";
}

int ConvolutionMOO::getW() {
  return this->videoCapter->getW();
}

int ConvolutionMOO::getH() {
  return this->videoCapter->getH();
}
