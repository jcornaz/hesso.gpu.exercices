#include "hip/hip_runtime.h"
#include "ConvolutionMOO.h"
#include "Device.h"
#include "OpencvTools.h"
#include "cudaType.h"
#include "ConvolutionConstants.h"

extern __global__ void convertInBlackAndWhite(uchar4* ptrDevPixels, int size);
extern __global__ void convolution(uchar4* ptrDevPixels, uchar4* ptrDevResult, int imageWidth, int imageHeight, float* ptrDevKernel, int kernelWidth, int kernelHeight);
extern __global__ void computeMinMax(uchar4* ptrDevPixels, int size, int* ptrDevMin, int* ptrDevMax);
extern __global__ void transform(uchar4* ptrDevPixels, int size, int* ptrDevBlack, int* ptrDevWhite);
extern float* getPtrDevKernel();

ConvolutionMOO::ConvolutionMOO(string videoPath, float* ptrKernel, int cudaGridDim, int cudaBlockDim) {

  this->dg = dim3(cudaGridDim, 1, 1);
  this->db = dim3(NB_THREADS_BY_BLOCK, 1, 1);
  Device::assertDim(dg, db);

  this->t = 0;
  this->videoCapter = new CVCaptureVideo("/media/Data/Video/autoroute.mp4");
  this->videoCapter->start();

  HANDLE_ERROR(hipMalloc(&this->ptrDevMin, sizeof(int)));
  HANDLE_ERROR(hipMalloc(&this->ptrDevMax, sizeof(int)));
  HANDLE_ERROR(hipMalloc(&this->ptrDevImage, sizeof(uchar4) * this->videoCapter->getW() * this->videoCapter->getH()));
  HANDLE_ERROR(hipMalloc(&this->ptrDevKernel, sizeof(float) * KERNEL_SIZE));
  HANDLE_ERROR(hipMemcpy(this->ptrDevKernel, ptrKernel, sizeof(float) * KERNEL_SIZE, hipMemcpyHostToDevice));
}

ConvolutionMOO::~ConvolutionMOO() {
  this->videoCapter->stop();
  free(this->videoCapter);
  HANDLE_ERROR(hipFree(this->ptrDevKernel));
  HANDLE_ERROR(hipFree(this->ptrDevImage));
  HANDLE_ERROR(hipFree(this->ptrDevMin));
  HANDLE_ERROR(hipFree(this->ptrDevMax));
}

/**
* Call periodicly by the api
*/
void ConvolutionMOO::process(uchar4* ptrDevPixels, int w, int h) {
  Mat matRGBA(h, w, CV_8UC4);
  Mat matBGR = this->videoCapter->provideBGR();
  OpencvTools::switchRB(matRGBA, matBGR);
  uchar4* ptrImage = OpencvTools::castToUchar4(matRGBA);
  int imageSize = w * h;

  HANDLE_ERROR(hipMemcpy(this->ptrDevImage, ptrImage, sizeof(uchar4) * imageSize, hipMemcpyHostToDevice));

  convertInBlackAndWhite<<<dg,db>>>(this->ptrDevImage, imageSize);
  convolution<<<dg,db>>>(this->ptrDevImage, ptrDevPixels, w, h, this->ptrDevKernel, KERNEL_WIDTH, KERNEL_WIDTH);
  computeMinMax<<<dg,db>>>(ptrDevPixels, imageSize, this->ptrDevMin, this->ptrDevMax);
  transform<<<dg,db>>>(ptrDevPixels, imageSize, this->ptrDevMax, this->ptrDevMin);
}

/**
* Call periodicly by the api
*/
void ConvolutionMOO::animationStep() {
  this->t++;
}

float ConvolutionMOO::getAnimationPara() {
  return this->t;
}

string ConvolutionMOO::getTitle() {
  return "Convolution";
}

int ConvolutionMOO::getW() {
  return this->videoCapter->getW();
}

int ConvolutionMOO::getH() {
  return this->videoCapter->getH();
}
