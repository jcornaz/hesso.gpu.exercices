#include "hip/hip_runtime.h"
#include <iostream>

#include "cudaTools.h"
#include "Device.h"

extern __global__ void computePIWithSlicing(float* ptrDevResult, int nbSlices);

bool isPIWithSlicingOk();

bool isPIWithSlicingOk() {

  dim3 dg = dim3(128, 1, 1);
  dim3 db = dim3(512, 1, 1);

  Device::assertDim(dg, db);

  float piValue;
  float* ptrDevResult;

  HANDLE_ERROR(hipMalloc(&ptrDevResult, sizeof(float)));
  HANDLE_ERROR(hipMemset(ptrDevResult, 0, sizeof(float)));

  computePIWithSlicing<<<dg,db>>>(ptrDevResult, 1000000);

  HANDLE_ERROR(hipMemcpy(&piValue, ptrDevResult, sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipFree(ptrDevResult));

  std::cout << "PI = " << piValue << " (with slicing)" << std::endl;

  return abs(piValue - 3.141592653589793f) < 0.0000000001;
}
