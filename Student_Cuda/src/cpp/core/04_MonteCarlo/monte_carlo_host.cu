#include "hip/hip_runtime.h"
#include <iostream>

#include "cudaTools.h"
#include "Device.h"

extern __global__ void computePIWithMonteCarlo(float* ptrDevResult, int nbSlices);

bool isMonteCarloOk();

bool isMonteCarloOk() {

  dim3 dg = dim3(128, 1, 1);
  dim3 db = dim3(512, 1, 1);

  Device::assertDim(dg, db);

  float piValue;
  float* ptrDevResult;

  HANDLE_ERROR(hipMalloc(&ptrDevResult, sizeof(float)));
  HANDLE_ERROR(hipMemset(ptrDevResult, 0, sizeof(float)));

  computePIWithMonteCarlo<<<dg,db>>>(ptrDevResult, 1000000);

  HANDLE_ERROR(hipMemcpy(&piValue, ptrDevResult, sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipFree(ptrDevResult));

  std::cout << "PI = " << piValue << " (with Monte Carlo)" << std::endl;

  return abs(piValue - 3.141592653589793f) < 0.001;
}
